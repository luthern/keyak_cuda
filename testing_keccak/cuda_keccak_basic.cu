#include "hip/hip_runtime.h"
/* Author: Noah Luther                                            */
/* Keccak-p permutation for Keyak authenticated cipher.           */
/* Based on implementation of Keccak-f by Gerhard Hoffman.        */
/**/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <inttypes.h>
#include <errno.h>
#include <hip/hip_runtime.h>
#include "cuda_keccak_basic.cuh"

static uint64_t *d_data;

#define ROUNDS        12
#define R64(a,b,c) (((a) << b) ^ ((a) >> c)) /* works on the GPU also for 
                                                b = 64 or c = 64 */
static const uint64_t round_const[5][ROUNDS] = {
    {0x000000008000808BULL, 0x800000000000008BULL, 0x8000000000008089ULL,
     0x8000000000008003ULL, 0x8000000000008002ULL, 0x8000000000000080ULL,
     0x000000000000800AULL, 0x800000008000000AULL, 0x8000000080008081ULL,
     0x8000000000008080ULL, 0x0000000080000001ULL, 0x8000000080008008ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL}};

/* Rho-Offsets. Note that for each entry pair their respective sum is 64.
   Only the first entry of each pair is a rho-offset. The second part is
   used in the R64 macros. */
static const uint8_t rho_offsets[25][2] = {
       /*y=0*/         /*y=1*/         /*y=2*/         /*y=3*/         /*y=4*/
/*x=0*/{ 0,64}, /*x=1*/{44,20}, /*x=2*/{43,21}, /*x=3*/{21,43}, /*x=4*/{14,50},
/*x=1*/{ 1,63}, /*x=2*/{ 6,58}, /*x=3*/{25,39}, /*x=4*/{ 8,56}, /*x=0*/{18,46},
/*x=2*/{62, 2}, /*x=3*/{55, 9}, /*x=4*/{39,25}, /*x=0*/{41,23}, /*x=1*/{ 2,62},
/*x=3*/{28,36}, /*x=4*/{20,44}, /*x=0*/{ 3,61}, /*x=1*/{45,19}, /*x=2*/{61, 3},
/*x=4*/{27,37}, /*x=0*/{36,28}, /*x=1*/{10,54}, /*x=2*/{15,49}, /*x=3*/{56, 8}};

static const uint8_t a_host[25] = {
    0,  6, 12, 18, 24,
    1,  7, 13, 19, 20,
    2,  8, 14, 15, 21,
    3,  9, 10, 16, 22,
    4,  5, 11, 17, 23};

static const uint8_t b_host[25] = {
    0,  1,  2,  3, 4,
    1,  2,  3,  4, 0,
    2,  3,  4,  0, 1,
    3,  4,  0,  1, 2,
    4,  0,  1,  2, 3};

static const uint8_t c_host[25][3] = {
    { 0, 1, 2}, { 1, 2, 3}, { 2, 3, 4}, { 3, 4, 0}, { 4, 0, 1},
    { 5, 6, 7}, { 6, 7, 8}, { 7, 8, 9}, { 8, 9, 5}, { 9, 5, 6},
    {10,11,12}, {11,12,13}, {12,13,14}, {13,14,10}, {14,10,11},
    {15,16,17}, {16,17,18}, {17,18,19}, {18,19,15}, {19,15,16},
    {20,21,22}, {21,22,23}, {22,23,24}, {23,24,20}, {24,20,21}};

static const uint8_t d_host[25] = {
          0,  1,  2,  3,  4,
         10, 11, 12, 13, 14,
         20, 21, 22, 23, 24,
          5,  6,  7,  8,  9,
         15, 16, 17, 18, 19};

__device__ __constant__ uint8_t a[25];
__device__ __constant__ uint8_t b[25];
__device__ __constant__ uint8_t c[25][3];
__device__ __constant__ uint8_t d[25];
__device__ __constant__ uint8_t ro[25][2];
__device__ __constant__ uint64_t rc[5][ROUNDS];

__global__
void keccak_p_kernel(uint64_t *data) {
    int const t = threadIdx.x;
    int const s = threadIdx.x%5;

    __shared__ uint64_t A[25];
    __shared__ uint64_t C[25];
    __shared__ uint64_t D[25];

    if (t < 25) {
        A[t] = data[t];

        for(int i=0;i<ROUNDS;++i) { 
            C[t] = A[s]^A[s+5]^A[s+10]^A[s+15]^A[s+20];
            D[t] = C[b[20+s]] ^ R64(C[b[5+s]],1,63);
            C[t] = R64(A[a[t]]^D[b[t]], ro[t][0], ro[t][1]);
            A[d[t]] = C[c[t][0]] ^ ((~C[c[t][1]]) & C[c[t][2]]); 
            A[t] ^= rc[(t==0) ? 0 : 1][i]; 
        }
    data[t] = A[t];
    }
}

void gpu_init_keccak_tables()
{
    /* copy the tables from host to GPU */
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), a_host, sizeof(a_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), b_host, sizeof(b_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c), c_host, sizeof(c_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d), d_host, sizeof(d_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ro), rho_offsets, sizeof(rho_offsets)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(rc), round_const, sizeof(round_const)));

    /* allocate space for the state on GPU */
    HANDLE_ERROR(hipMalloc((void **)&d_data, 200));
}

/* Modifies state with 12 rounds of Keccak.
   Uses the LFSR round constants for Keyak.
*/
void call_keccak_basic_kernel(uint64_t * state) {

    /* copy the data from the state to the GPU */
    HANDLE_ERROR(hipMemcpy(d_data, state, 200, hipMemcpyHostToDevice));

    /* permute the state */
    keccak_p_kernel<<<1,32>>>(d_data);

    /* fetch the generated data */
    HANDLE_ERROR(hipMemcpy(state, d_data, 200, hipMemcpyDeviceToHost));
}

void cleanup_state()
{
    /* clean up the tables on the GPU */
    HANDLE_ERROR(hipFree(d_data));
}

void _HANDLE_ERROR(hipError_t e, int line)
{
    if (e != hipSuccess)
    {
        printf("line: %d. error %s\n", line, hipGetErrorString(e));
        exit(1);
    }
}
