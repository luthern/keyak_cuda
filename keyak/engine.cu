#include <string.h>
#include <assert.h>
#include "engine.h"
#include "misc.h"
#include "piston.h"

#include "utils.h"

// cuda does not support external linkage
#include "piston.cu"

void engine_init(Engine * e, Piston * pistons)
{
    memset(e,0,sizeof(Engine));
    e->pistons = pistons;
    e->phase = EngineFresh;


    // TODO consider making this one contiguous block
    HANDLE_ERROR(hipMalloc(&e->p_in, PISTON_RS * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMalloc(&e->p_out, PISTON_RS * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMalloc(&e->p_state, PISTON_RS * KEYAK_F_WIDTH / 8 ));

    HANDLE_ERROR(hipMemset(e->p_state,0,PISTON_RS * KEYAK_F_WIDTH / 8 ));
}

void engine_destroy(Engine * e)
{
    hipFree(e->p_in);
    hipFree(e->p_out);
}

void engine_restart(Engine * e)
{
    e->phase = EngineFresh;
}

void engine_spark(Engine * e, uint8_t eom, uint8_t * offsets)
{
    uint8_t i;
    for (i=0; i < KEYAK_NUM_PISTONS; i++)
    {
        piston_spark(&e->pistons[i],eom, offsets[i]);
    }
    memmove(e->Et, offsets, sizeof(uint8_t)*KEYAK_NUM_PISTONS);
}

void engine_get_tags(Engine * e, Buffer * T, uint8_t * L)
{
    assert(e->phase == EngineEndOfMessage);
    engine_spark(e, 1, L);
    uint8_t i;
    for (i = 0; i < KEYAK_NUM_PISTONS; i++)
    {
        piston_get_tag(&e->pistons[i], T, L[i]);
    }
    e->phase = EngineFresh;
}

uint8_t offsets_zero[KEYAK_NUM_PISTONS];
void engine_precompute()
{
    memset(offsets_zero, 0, sizeof(offsets_zero));

}

void engine_inject(Engine * e, Buffer * A)
{    
    assert(
            e->phase == EngineCrypted ||
            e->phase == EngineEndOfCrypt ||
            e->phase == EngineFresh
            );
    uint8_t cryptingFlag = (
            e->phase == EngineCrypted ||
            e->phase == EngineEndOfCrypt
            );

    uint8_t i;
    for(i=0; i < KEYAK_NUM_PISTONS; i++)
    {
        piston_inject(&e->pistons[i],A,cryptingFlag);
    }
    if (e->phase == EngineCrypted || buffer_has_more(A))
    {
        engine_spark(e,0, offsets_zero);
        e->phase = EngineFresh;
    }
    else
    {
        e->phase = EngineEndOfMessage;
    }
}

void engine_inject_collective(Engine * e, Buffer * X, uint8_t dFlag)
{
    assert(e->phase == EngineFresh);
    Buffer Xt[KEYAK_NUM_PISTONS];
    uint8_t i;
    for (i=0; i< KEYAK_NUM_PISTONS; i++)
    {
        buffer_init(Xt+i, NULL, 0);
    }

    while(buffer_has_more(X))
    {
        uint8_t b = buffer_get(X); 
        for (i=0; i< KEYAK_NUM_PISTONS; i++)
        {
            buffer_put(&Xt[i],b);
        }
    }

    if (dFlag)
    {
        for (i=0; i< KEYAK_NUM_PISTONS; i++)
        {
            buffer_put(&Xt[i],KEYAK_NUM_PISTONS);
            buffer_put(&Xt[i],i);
        }
    }
    for (i=0; i< KEYAK_NUM_PISTONS; i++)
    {
        buffer_seek(&Xt[i],0);
    }

    while(buffer_has_more(Xt))
    {
        for (i=0; i< KEYAK_NUM_PISTONS; i++)
        {
            piston_inject(&e->pistons[i], &Xt[i], 0);
        }
        if (buffer_has_more(Xt))
        {
            uint8_t offsets[KEYAK_NUM_PISTONS];
            memset(offsets, 0, sizeof(offsets));
            engine_spark(e, 0, offsets);
        }
    }

    e->phase = EngineEndOfMessage;
}


static int _init_crypt = 0;
void engine_crypt(Engine * e, Buffer * I, Buffer * O, uint8_t unwrapFlag)
{
    assert(e->phase == EngineFresh);

    printf("start: %d  end: %d  leftover %d\n",
            I->offset, I->length, I->length - I->offset);
    printf("the total i can saturate is %d\n",
            PISTON_RS * KEYAK_NUM_PISTONS);

    uint32_t amt = MIN(PISTON_RS*KEYAK_NUM_PISTONS, I->length - I->offset);

    printf("plain text 1:\n");
    dump_hex(I->buf, amt);

    // TODO consider copying more than 1 block
    // Copy block of input to GPU
    HANDLE_ERROR(hipMemcpy(e->p_in,I->buf + I->offset,
                amt,
                hipMemcpyHostToDevice));

    // use minimum number of blocks to get threads we need.
    uint32_t numBlocks = amt / MAX_CUDA_THREADS_PER_BLOCK + 1;

    piston_crypt<<<numBlocks,MAX_CUDA_THREADS_PER_BLOCK>>>(e->p_in,e->p_out,e->p_state,amt, unwrapFlag);

    // Copy the output of pistons
    HANDLE_ERROR(hipMemcpy(O->buf, e->p_out,
                amt,
                hipMemcpyDeviceToHost));

    printf("cipher text 1:\n");
    dump_hex(O->buf, amt);

    I->offset += amt;

    e->phase = buffer_has_more(I) ? EngineCrypted : EngineEndOfCrypt;

    exit(1);
}


