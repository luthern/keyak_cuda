#include <string.h>
#include <assert.h>
#include "engine.h"
#include "misc.h"
#include "piston.h"

#include "utils.h"

// cuda does not support external linkage
#include "keccak.cu"
#include "piston.cu"

#if 1

void dump_state(Engine * e, int piston)
{
    uint8_t tmp[KEYAK_STATE_SIZE];

    HANDLE_ERROR(hipMemcpy(tmp,e->p_state + piston * KEYAK_STATE_SIZE,
                            KEYAK_STATE_SIZE, hipMemcpyDeviceToHost));
    dump_hex(tmp, sizeof(tmp));
}

#endif


void engine_init(Engine * e, Piston * pistons)
{
    memset(e,0,sizeof(Engine));
    e->pistons = pistons;
    e->phase = EngineFresh;

    // TODO consider making these all one contiguous block or even different memories
    HANDLE_ERROR(hipMalloc(&e->p_in, PISTON_RS * KEYAK_NUM_PISTONS ));

    HANDLE_ERROR(hipMalloc(&e->p_out, PISTON_RS * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMalloc(&e->p_state, KEYAK_STATE_SIZE * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMalloc(&e->p_tmp, KEYAK_BUFFER_SIZE * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMalloc(&e->p_offsets, KEYAK_NUM_PISTONS ));

    HANDLE_ERROR(hipMemset(e->p_state,0, KEYAK_STATE_SIZE * KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMemset(e->p_offsets,0,KEYAK_NUM_PISTONS ));
    HANDLE_ERROR(hipMemset(e->p_tmp,0,KEYAK_BUFFER_SIZE * KEYAK_NUM_PISTONS ));
}

void engine_destroy(Engine * e)
{
    printf("engine_destroyed\n");
    hipFree(e->p_in);
    hipFree(e->p_out);
}

void engine_restart(Engine * e)
{
    e->phase = EngineFresh;
}

void engine_spark(Engine * e, uint8_t eom, uint8_t * offsets)
{
    //printf("ENGINE_SPARK\n");

    // TODO pass offsets array
    hipMemcpy(e->p_offsets, offsets, KEYAK_NUM_PISTONS, hipMemcpyHostToDevice);

    //printf("spark state 1 : \n");
    //int j;
    //for (j=0; j < KEYAK_NUM_PISTONS; j++)
    //{
    //    printf("piston %d\n", j);
    //    dump_state(e,j);
    ///}


    piston_spark<<<KEYAK_NUM_PISTONS,1>>>
        (e->p_state, eom, e->p_offsets);

/*    printf("spark state 2 : \n");*/
    /*for (j=0; j < KEYAK_NUM_PISTONS; j++)*/
    /*{*/
        /*printf("piston %d\n", j);*/
        /*dump_state(e,j);*/
    /*}*/


    memmove(e->Et, offsets, KEYAK_NUM_PISTONS);
}

void engine_get_tags(Engine * e, Buffer * T, uint8_t * L)
{
    assert(e->phase == EngineEndOfMessage);
    uint8_t i;
/*    printf("get tags state 1: \n");*/
    /*for (j=0; j < KEYAK_NUM_PISTONS; j++)*/
    /*{*/
        /*printf("piston %d\n", j);*/
        /*dump_state(e,j);*/
    /*}*/
    engine_spark(e, 1, L);

  /*  printf("get tags state 2: \n");*/
    /*for (j=0; j < KEYAK_NUM_PISTONS; j++)*/
    /*{*/
        /*printf("piston %d\n", j);*/
        /*dump_state(e,j);*/
    /*}*/


    for (i = 0; i < KEYAK_NUM_PISTONS; i++)
    {
        if (L[i])
        {
            // TODO consider making one copy or making this async
            assert(L[i] <= PISTON_RS);
            HANDLE_ERROR(
                    hipMemcpy(T->buf + T->length,
                                e->p_state + i * KEYAK_STATE_SIZE,
                                L[i], hipMemcpyDeviceToHost)
                    );
            /*printf("copied tag bytes %d:\n",i);*/
            /*dump_hex(T->buf + T->length, L[i]);*/

            T->length += L[i];
        }
    }
    e->phase = EngineFresh;
}

uint8_t offsets_zero[KEYAK_NUM_PISTONS];
void engine_precompute()
{
    memset(offsets_zero, 0, sizeof(offsets_zero));
}

void engine_inject(Engine * e, Buffer * A)
{
    //printf("ENGINE_INJECT\n");
    assert(
            e->phase == EngineCrypted ||
            e->phase == EngineEndOfCrypt ||
            e->phase == EngineFresh
            );
    uint8_t cryptingFlag = (
            e->phase == EngineCrypted ||
            e->phase == EngineEndOfCrypt
            );

    uint32_t amt = MIN(PISTON_RA*KEYAK_NUM_PISTONS, A->length - A->offset);
    uint8_t i;

    HANDLE_ERROR(
            hipMemcpy(e->p_tmp, A->buf + A->offset, amt, hipMemcpyHostToDevice)
            );

    //printf("injecting %d bytes\n", amt);
    
    //printf("inject state 1 : \n");
    //int j;
    //for (j=0; j < KEYAK_NUM_PISTONS; j++)
   // {
    //    printf("piston %d\n", j);
    //    dump_state(e,j);
    //}


    piston_inject_seq<<<KEYAK_NUM_PISTONS, PISTON_RA>>>
        (e->p_state, e->p_tmp, 0, amt, cryptingFlag);
    A->offset += amt;

//    printf("inject state 2 : \n");
//    for (j=0; j < KEYAK_NUM_PISTONS; j++)
//    {
//        printf("piston %d\n", j);
//        dump_state(e,j);
//    }

    if (e->phase == EngineCrypted || buffer_has_more(A))
    {
        engine_spark(e,0, offsets_zero);
        e->phase = EngineFresh;
    }
    else
    {
        e->phase = EngineEndOfMessage;
    }
}

#if 0
static void dump_tmp_buf(Engine * e)
{
    uint8_t tmp[KEYAK_BUFFER_SIZE*KEYAK_NUM_PISTONS];

    HANDLE_ERROR(hipMemcpy(tmp, e->p_tmp, sizeof(tmp),
                hipMemcpyDeviceToHost));

    int offset = 0;
    int i;
    for (i=0; i < KEYAK_NUM_PISTONS; i++)
    {
        dump_hex(tmp + offset * KEYAK_BUFFER_SIZE, 100);
        printf("\r\n");
    }
}

static void dump_hash(Engine * e, int piston)
{
    uint8_t tmp[KEYAK_STATE_SIZE];

    HANDLE_ERROR(hipMemcpy(tmp,e->p_state + piston * KEYAK_STATE_SIZE,
                            KEYAK_STATE_SIZE, hipMemcpyDeviceToHost));
    PERMUTE(tmp);
    dump_hex(tmp, sizeof(tmp));
}
#endif



void engine_inject_collective(Engine * e, Buffer * X, uint8_t dFlag)
{
    assert(e->phase == EngineFresh);
    //printf("ENGINE_INJECT_COLLECTIVE\n");

    /*printf("collectively injecting %d bytes\n", X->length);*/

    /*dump_hex(X->buf, X->length);*/

    /*printf("COLLECTIVE INPUT STATE :\n");*/
    /*int j;*/
    /*for (j=0; j < KEYAK_NUM_PISTONS; j++)*/
    /*{*/
        /*[>printf("piston %d\n", j);<]*/
        /*dump_state(e,j);*/
    /*}*/



    if (dFlag)
    {
        /*printf("diversivefying\n");*/
        buffer_put(X,KEYAK_NUM_PISTONS);
        buffer_put(X,0);
    }


    // TODO should support variable length
    assert(X->length < KEYAK_BUFFER_SIZE);

    // copy collective to gpu
    HANDLE_ERROR(hipMemcpy(e->p_tmp,X->buf,
                X->length,
                hipMemcpyHostToDevice));

    // TODO check if its just better to make 8 copies
    // but i think device to device copying would be speedier than
    // host to device cuz pci bus
    dup_for_pistons<<<1,KEYAK_NUM_PISTONS>>>(e->p_tmp, X->length,dFlag);


    uint32_t i;
    for (i=0; i < X->length; i += PISTON_RA)
    {
        if ( i + PISTON_RA >= X->length)
        {
            /*printf("injecting %d bytes\n", X->length - i);*/
            piston_inject_uniform<<<KEYAK_NUM_PISTONS, PISTON_RA>>>(e->p_state,
                    e->p_tmp, i, X->length - i, 0);
        }
        else
        {
            /*printf("injecting PISTON_RA bytes\n");*/
            piston_inject_uniform<<<KEYAK_NUM_PISTONS, PISTON_RA>>>(e->p_state,
                    e->p_tmp, i, PISTON_RA, 0);
            // data dependency
            piston_spark<<<KEYAK_NUM_PISTONS,1>>>
                (e->p_state, 0, NULL);

        }

/*        printf("COLLECTIVE INJECT STATE :\n");*/
        /*int j;*/
        /*for (j=0; j < KEYAK_NUM_PISTONS; j++)*/
        /*{*/
            /*printf("piston %d\n", j);*/
            /*dump_state(e,j);*/
        /*}*/



        // test
    }




    e->phase = EngineEndOfMessage;
}

//static int iter =0 ;
void engine_crypt(Engine * e, Buffer * I, Buffer * O, uint8_t unwrapFlag)
{

    assert(e->phase == EngineFresh);
    uint32_t amt = MIN(PISTON_RS*KEYAK_NUM_PISTONS, I->length - I->offset);

    //printf("state: \n");
    //int j;
    //for (j=0; j < KEYAK_NUM_PISTONS; j++)
    //{
    //    printf("piston %d\n", j);
    //    dump_state(e,j);
    //}

    //printf("plain text %d (offset: %d):\n",iter, I->offset);
    //dump_hex(I->buf, amt);

    // TODO consider copying more than 1 block
    // Copy block of input to GPU
    HANDLE_ERROR(hipMemcpy(e->p_in,I->buf + I->offset,
                amt,
                hipMemcpyHostToDevice));
    
    piston_crypt<<<KEYAK_NUM_PISTONS,PISTON_RS>>>
        (e->p_in,e->p_out,e->p_state,amt, unwrapFlag);

    // Copy the output of pistons
    assert(O->length + amt < KEYAK_BUFFER_SIZE);
    HANDLE_ERROR(hipMemcpy(O->buf + O->length, e->p_out,
                amt,
                hipMemcpyDeviceToHost));
    //printf("cipher text %d:\n",iter++);
    //dump_hex(O->buf + O->length, amt);

    O->length += amt;
    I->offset += amt;

    e->phase = buffer_has_more(I) ? EngineCrypted : EngineEndOfCrypt;

}


