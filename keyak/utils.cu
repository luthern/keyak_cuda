#include <hip/hip_runtime.h>
#include "utils.h"

void _HANDLE_ERROR(hipError_t e, int line)
{
    if (e != hipSuccess)
    {
        printf("line: %d. error %s\n", line, hipGetErrorString(e));
        exit (1);
    }
}
