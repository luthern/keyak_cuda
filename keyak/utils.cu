#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.h"

void _HANDLE_ERROR(hipError_t e, const char * file, int line)
{
    if (e != hipSuccess)
    {
        printf("%s: %d. error %s\n", file, line, hipGetErrorString(e));
        exit (1);
    }
}
