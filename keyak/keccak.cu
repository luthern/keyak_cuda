#include "hip/hip_runtime.h"
/* Author: Noah Luther                                            */
/* Keccak-p permutation for Keyak authenticated cipher.           */
/* Based on implementation of Keccak-f by Gerhard Hoffman.        */
/**/

#define IS_THIS_WORKING
#ifdef IS_THIS_WORKING

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <inttypes.h>
#include <errno.h>
#include <hip/hip_runtime.h>

static uint64_t *d_data;

#define ROUNDS        12
#define R64(a,b,c) (((a) << b) ^ ((a) >> c)) /* works on the GPU also for 
                                                b = 64 or c = 64 */
static const uint64_t round_const[5][ROUNDS] = {
    {0x000000008000808BULL, 0x800000000000008BULL, 0x8000000000008089ULL,
     0x8000000000008003ULL, 0x8000000000008002ULL, 0x8000000000000080ULL,
     0x000000000000800AULL, 0x800000008000000AULL, 0x8000000080008081ULL,
     0x8000000000008080ULL, 0x0000000080000001ULL, 0x8000000080008008ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL},
    {0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL,
     0ULL, 0ULL, 0ULL, 0ULL, 0ULL, 0ULL}};

/* Rho-Offsets. Note that for each entry pair their respective sum is 64.
   Only the first entry of each pair is a rho-offset. The second part is
   used in the R64 macros. */
static const uint8_t rho_offsets[25][2] = {
       /*y=0*/         /*y=1*/         /*y=2*/         /*y=3*/         /*y=4*/
/*x=0*/{ 0,64}, /*x=1*/{44,20}, /*x=2*/{43,21}, /*x=3*/{21,43}, /*x=4*/{14,50},
/*x=1*/{ 1,63}, /*x=2*/{ 6,58}, /*x=3*/{25,39}, /*x=4*/{ 8,56}, /*x=0*/{18,46},
/*x=2*/{62, 2}, /*x=3*/{55, 9}, /*x=4*/{39,25}, /*x=0*/{41,23}, /*x=1*/{ 2,62},
/*x=3*/{28,36}, /*x=4*/{20,44}, /*x=0*/{ 3,61}, /*x=1*/{45,19}, /*x=2*/{61, 3},
/*x=4*/{27,37}, /*x=0*/{36,28}, /*x=1*/{10,54}, /*x=2*/{15,49}, /*x=3*/{56, 8}};

static const uint8_t a_host[25] = {
    0,  6, 12, 18, 24,
    1,  7, 13, 19, 20,
    2,  8, 14, 15, 21,
    3,  9, 10, 16, 22,
    4,  5, 11, 17, 23};

static const uint8_t b_host[25] = {
    0,  1,  2,  3, 4,
    1,  2,  3,  4, 0,
    2,  3,  4,  0, 1,
    3,  4,  0,  1, 2,
    4,  0,  1,  2, 3};

static const uint8_t c_host[25][3] = {
    { 0, 1, 2}, { 1, 2, 3}, { 2, 3, 4}, { 3, 4, 0}, { 4, 0, 1},
    { 5, 6, 7}, { 6, 7, 8}, { 7, 8, 9}, { 8, 9, 5}, { 9, 5, 6},
    {10,11,12}, {11,12,13}, {12,13,14}, {13,14,10}, {14,10,11},
    {15,16,17}, {16,17,18}, {17,18,19}, {18,19,15}, {19,15,16},
    {20,21,22}, {21,22,23}, {22,23,24}, {23,24,20}, {24,20,21}};

static const uint8_t d_host[25] = {
          0,  1,  2,  3,  4,
         10, 11, 12, 13, 14,
         20, 21, 22, 23, 24,
          5,  6,  7,  8,  9,
         15, 16, 17, 18, 19};

__device__ __constant__ uint8_t a[25];
__device__ __constant__ uint8_t b[25];
__device__ __constant__ uint8_t c[25][3];
__device__ __constant__ uint8_t d[25];
__device__ __constant__ uint8_t ro[25][2];
__device__ __constant__ uint64_t rc[5][ROUNDS];

__device__
void keccak_p_kernel(uint64_t *data) {
    int const t = threadIdx.x;
    int const s = threadIdx.x%5;

    __shared__ uint64_t A[25];
    __shared__ uint64_t C[25];
    __shared__ uint64_t D[25];

    if (t < 25) {
        A[t] = data[t];

        for(int i=0;i<ROUNDS;++i) { 
            C[t] = A[s]^A[s+5]^A[s+10]^A[s+15]^A[s+20];
            D[t] = C[b[20+s]] ^ R64(C[b[5+s]],1,63);
            C[t] = R64(A[a[t]]^D[b[t]], ro[t][0], ro[t][1]);
            A[d[t]] = C[c[t][0]] ^ ((~C[c[t][1]]) & C[c[t][2]]); 
            A[t] ^= rc[(t==0) ? 0 : 1][i]; 
        }
    data[t] = A[t];
    }
}

void gpu_init_keccak_tables()
{
    /* copy the tables from host to GPU */
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(a), a_host, sizeof(a_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(b), b_host, sizeof(b_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c), c_host, sizeof(c_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d), d_host, sizeof(d_host)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ro), rho_offsets, sizeof(rho_offsets)));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(rc), round_const, sizeof(round_const)));

    /* allocate space for the state on GPU */
    HANDLE_ERROR(hipMalloc((void **)&d_data, 200));
}

/* Modifies state with 12 rounds of Keccak.
   Uses the LFSR round constants for Keyak.
*/
/*void call_keccak_basic_kernel(uint64_t * state) {*/

    /*[> copy the data from the state to the GPU <]*/
    /*HANDLE_ERROR(hipMemcpy(d_data, state, 200, hipMemcpyHostToDevice));*/

    /*[> permute the state <]*/
    /*keccak_p_kernel<<<1,32>>>(d_data);*/

    /*[> fetch the generated data <]*/
    /*HANDLE_ERROR(hipMemcpy(state, d_data, 200, hipMemcpyDeviceToHost));*/
/*}*/

void cleanup_state()
{
    /* clean up the tables on the GPU */
    HANDLE_ERROR(hipFree(d_data));
}

void _HANDLE_ERROR(hipError_t e, int line)
{
    if (e != hipSuccess)
    {
        printf("line: %d. error %s\n", line, hipGetErrorString(e));
        exit(1);
    }
}
#else

#include "keccak.h"

#define     UNROLL_CHILOOP


#if defined(__GNUC__)
#define ALIGN __attribute__ ((aligned(32)))
#elif defined(_MSC_VER)
#define ALIGN __declspec(align(32))
#else
#define ALIGN
#endif

#if defined(_MSC_VER)
#define ROL64(a, offset) _rotl64(a, offset)
#elif defined(UseSHLD)
    #define ROL64(x,N) ({ \
    register UINT64 __out; \
    register UINT64 __in = x; \
    __asm__ ("shld %2,%0,%0" : "=r"(__out) : "0"(__in), "i"(N)); \
    __out; \
    })
#else
#define ROL64(a, offset) ((((UINT64)a) << offset) ^ (((UINT64)a) >> (64-offset)))
#endif

__host__ __device__ static tKeccakLane KeccakF1600_GetNextRoundConstant( UINT8 *LFSR )
{
    tSmallUInt i;
    tKeccakLane    roundConstant;
    tSmallUInt doXOR;
    tSmallUInt tempLSFR;

    roundConstant = 0;
    tempLSFR = *LFSR;
    for(i=1; i<128; i <<= 1)
    {
        doXOR = tempLSFR & 1;
        if ((tempLSFR & 0x80) != 0)
            // Primitive polynomial over GF(2): x^8+x^6+x^5+x^4+1
            tempLSFR = (tempLSFR << 1) ^ 0x71;
        else
            tempLSFR <<= 1;

        if ( doXOR != 0 )
            roundConstant ^= (tKeccakLane)1ULL << (i - 1);
    }
    *LFSR = (UINT8)tempLSFR;
    return ( roundConstant );
}

__host__ __device__ void KeccakP1600_StatePermute(void *argState, UINT8 rounds, UINT8 LFSRinitialState)
{
    const UINT8 KeccakF_RotationConstants[25] =
    {
        1,  3,  6, 10, 15, 21, 28, 36, 45, 55,  2, 14, 27, 41, 56,  8, 25, 43, 62, 18, 39, 61, 20, 44
    };

    const UINT8 KeccakF_PiLane[25] =
    {
        10,  7, 11, 17, 18,  3,  5, 16,  8, 21, 24,  4, 15, 23, 19, 13, 12,  2, 20, 14, 22,  9,  6,  1
    };

    //#define DIVISION_INSTRUCTION
#if    defined(DIVISION_INSTRUCTION)
#define    MOD5(argValue)    ((argValue) % 5)
#else
    const UINT8 KeccakF_Mod5[10] =
    {
        0, 1, 2, 3, 4, 0, 1, 2, 3, 4
    };
#define    MOD5(argValue)    KeccakF_Mod5[argValue]
#endif


    tSmallUInt x, y, round;
    tKeccakLane        temp;
    tKeccakLane        BC[5];
    tKeccakLane     *state;
    UINT8           LFSRstate;

    state = (tKeccakLane*)argState;
    LFSRstate = LFSRinitialState;
    round = rounds;
    do
    {
        // Theta
        for ( x = 0; x < 5; ++x )
        {
            BC[x] = state[x] ^ state[5 + x] ^ state[10 + x] ^ state[15 + x] ^ state[20 + x];
        }
        for ( x = 0; x < 5; ++x )
        {
            temp = BC[MOD5(x+4)] ^ ROL64(BC[MOD5(x+1)], 1);
            for ( y = 0; y < 25; y += 5 )
            {
                state[y + x] ^= temp;
            }
        }

        // Rho Pi
        temp = state[1];
        for ( x = 0; x < 24; ++x )
        {
            BC[0] = state[KeccakF_PiLane[x]];
            state[KeccakF_PiLane[x]] = ROL64( temp, KeccakF_RotationConstants[x] );
            temp = BC[0];
        }

        //    Chi
        for ( y = 0; y < 25; y += 5 )
        {
#if defined(UNROLL_CHILOOP)
            BC[0] = state[y + 0];
            BC[1] = state[y + 1];
            BC[2] = state[y + 2];
            BC[3] = state[y + 3];
            BC[4] = state[y + 4];
#else
            for ( x = 0; x < 5; ++x )
            {
                BC[x] = state[y + x];
            }
#endif
            for ( x = 0; x < 5; ++x )
            {
                state[y + x] = BC[x] ^((~BC[MOD5(x+1)]) & BC[MOD5(x+2)]);
            }
        }

        //    Iota
        state[0] ^= KeccakF1600_GetNextRoundConstant(&LFSRstate);
    }
    while( --round != 0 );
}

void gpu_init_keccak_tables()
{
}

#endif

