#include "hip/hip_runtime.h"
#include "keccak.h"

#define     UNROLL_CHILOOP


#if defined(__GNUC__)
#define ALIGN __attribute__ ((aligned(32)))
#elif defined(_MSC_VER)
#define ALIGN __declspec(align(32))
#else
#define ALIGN
#endif

#if defined(_MSC_VER)
#define ROL64(a, offset) _rotl64(a, offset)
#elif defined(UseSHLD)
    #define ROL64(x,N) ({ \
    register UINT64 __out; \
    register UINT64 __in = x; \
    __asm__ ("shld %2,%0,%0" : "=r"(__out) : "0"(__in), "i"(N)); \
    __out; \
    })
#else
#define ROL64(a, offset) ((((UINT64)a) << offset) ^ (((UINT64)a) >> (64-offset)))
#endif

__host__ __device__ static tKeccakLane KeccakF1600_GetNextRoundConstant( UINT8 *LFSR )
{
    tSmallUInt i;
    tKeccakLane    roundConstant;
    tSmallUInt doXOR;
    tSmallUInt tempLSFR;

    roundConstant = 0;
    tempLSFR = *LFSR;
    for(i=1; i<128; i <<= 1)
    {
        doXOR = tempLSFR & 1;
        if ((tempLSFR & 0x80) != 0)
            // Primitive polynomial over GF(2): x^8+x^6+x^5+x^4+1
            tempLSFR = (tempLSFR << 1) ^ 0x71;
        else
            tempLSFR <<= 1;

        if ( doXOR != 0 )
            roundConstant ^= (tKeccakLane)1ULL << (i - 1);
    }
    *LFSR = (UINT8)tempLSFR;
    return ( roundConstant );
}

__host__ __device__ void KeccakP1600_StatePermute(void *argState, UINT8 rounds, UINT8 LFSRinitialState)
{
    const UINT8 KeccakF_RotationConstants[25] =
    {
        1,  3,  6, 10, 15, 21, 28, 36, 45, 55,  2, 14, 27, 41, 56,  8, 25, 43, 62, 18, 39, 61, 20, 44
    };

    const UINT8 KeccakF_PiLane[25] =
    {
        10,  7, 11, 17, 18,  3,  5, 16,  8, 21, 24,  4, 15, 23, 19, 13, 12,  2, 20, 14, 22,  9,  6,  1
    };

    //#define DIVISION_INSTRUCTION
#if    defined(DIVISION_INSTRUCTION)
#define    MOD5(argValue)    ((argValue) % 5)
#else
    const UINT8 KeccakF_Mod5[10] =
    {
        0, 1, 2, 3, 4, 0, 1, 2, 3, 4
    };
#define    MOD5(argValue)    KeccakF_Mod5[argValue]
#endif


    tSmallUInt x, y, round;
    tKeccakLane        temp;
    tKeccakLane        BC[5];
    tKeccakLane     *state;
    UINT8           LFSRstate;

    state = (tKeccakLane*)argState;
    LFSRstate = LFSRinitialState;
    round = rounds;
    do
    {
        // Theta
        for ( x = 0; x < 5; ++x )
        {
            BC[x] = state[x] ^ state[5 + x] ^ state[10 + x] ^ state[15 + x] ^ state[20 + x];
        }
        for ( x = 0; x < 5; ++x )
        {
            temp = BC[MOD5(x+4)] ^ ROL64(BC[MOD5(x+1)], 1);
            for ( y = 0; y < 25; y += 5 )
            {
                state[y + x] ^= temp;
            }
        }

        // Rho Pi
        temp = state[1];
        for ( x = 0; x < 24; ++x )
        {
            BC[0] = state[KeccakF_PiLane[x]];
            state[KeccakF_PiLane[x]] = ROL64( temp, KeccakF_RotationConstants[x] );
            temp = BC[0];
        }

        //    Chi
        for ( y = 0; y < 25; y += 5 )
        {
#if defined(UNROLL_CHILOOP)
            BC[0] = state[y + 0];
            BC[1] = state[y + 1];
            BC[2] = state[y + 2];
            BC[3] = state[y + 3];
            BC[4] = state[y + 4];
#else
            for ( x = 0; x < 5; ++x )
            {
                BC[x] = state[y + x];
            }
#endif
            for ( x = 0; x < 5; ++x )
            {
                state[y + x] = BC[x] ^((~BC[MOD5(x+1)]) & BC[MOD5(x+2)]);
            }
        }

        //    Iota
        state[0] ^= KeccakF1600_GetNextRoundConstant(&LFSRstate);
    }
    while( --round != 0 );
}

